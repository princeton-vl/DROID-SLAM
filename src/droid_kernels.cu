#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>

#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Parallel.h>

// #include "utils.cuh"

#include <Eigen/Sparse>
#include <Eigen/SparseCore>
#include <Eigen/SparseCholesky>

#ifdef _WIN32
    #include <cstdint>
    typedef int64_t LongType;
#else
    typedef long LongType;
#endif

typedef Eigen::SparseMatrix<double> SpMat;
typedef Eigen::Triplet<double> T;
typedef std::vector<std::vector<LongType>> graph_t;
typedef std::vector<torch::Tensor> tensor_list_t;



#define MIN_DEPTH 0.25

#define THREADS 256
#define NUM_BLOCKS(batch_size) ((batch_size + THREADS - 1) / THREADS)


#define GPU_1D_KERNEL_LOOP(k, n) \
  for (size_t k = threadIdx.x; k<n; k += blockDim.x)


__device__ void warpReduce(volatile float *sdata, unsigned int tid) {
  sdata[tid] += sdata[tid + 32];
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid +  8];
  sdata[tid] += sdata[tid +  4];
  sdata[tid] += sdata[tid +  2];
  sdata[tid] += sdata[tid +  1];
}

__device__ void blockReduce(volatile float *sdata) {
  unsigned int tid = threadIdx.x;
  __syncthreads();

  // if (threadIdx.x < 256) {sdata[tid] += sdata[tid + 256]; } __syncthreads();
  if (threadIdx.x < 128) {sdata[tid] += sdata[tid + 128]; } __syncthreads();
  if (threadIdx.x <  64) {sdata[tid] += sdata[tid +  64]; } __syncthreads();

  if (tid < 32) warpReduce(sdata, tid);
  __syncthreads();
}


__device__ void
actSO3(const float *q, const float *X, float *Y) {
  float uv[3];
  uv[0] = 2.0 * (q[1]*X[2] - q[2]*X[1]);
  uv[1] = 2.0 * (q[2]*X[0] - q[0]*X[2]);
  uv[2] = 2.0 * (q[0]*X[1] - q[1]*X[0]);

  Y[0] = X[0] + q[3]*uv[0] + (q[1]*uv[2] - q[2]*uv[1]);
  Y[1] = X[1] + q[3]*uv[1] + (q[2]*uv[0] - q[0]*uv[2]);
  Y[2] = X[2] + q[3]*uv[2] + (q[0]*uv[1] - q[1]*uv[0]);
}

__device__  void
actSE3(const float *t, const float *q, const float *X, float *Y) {
  actSO3(q, X, Y);
  Y[3] = X[3];
  Y[0] += X[3] * t[0];
  Y[1] += X[3] * t[1];
  Y[2] += X[3] * t[2];
}

__device__ void
adjSE3(const float *t, const float *q, const float *X, float *Y) {
  float qinv[4] = {-q[0], -q[1], -q[2], q[3]};
  actSO3(qinv, &X[0], &Y[0]);
  actSO3(qinv, &X[3], &Y[3]);

  float u[3], v[3];
  u[0] = t[2]*X[1] - t[1]*X[2];
  u[1] = t[0]*X[2] - t[2]*X[0];
  u[2] = t[1]*X[0] - t[0]*X[1];

  actSO3(qinv, u, v);
  Y[3] += v[0];
  Y[4] += v[1];
  Y[5] += v[2];
}

__device__ void 
relSE3(const float *ti, const float *qi, const float *tj, const float *qj, float *tij, float *qij) {
  qij[0] = -qj[3] * qi[0] + qj[0] * qi[3] - qj[1] * qi[2] + qj[2] * qi[1],
  qij[1] = -qj[3] * qi[1] + qj[1] * qi[3] - qj[2] * qi[0] + qj[0] * qi[2],
  qij[2] = -qj[3] * qi[2] + qj[2] * qi[3] - qj[0] * qi[1] + qj[1] * qi[0],
  qij[3] =  qj[3] * qi[3] + qj[0] * qi[0] + qj[1] * qi[1] + qj[2] * qi[2],

  actSO3(qij, ti, tij);
  tij[0] = tj[0] - tij[0];
  tij[1] = tj[1] - tij[1];
  tij[2] = tj[2] - tij[2];
}

  
__device__ void
expSO3(const float *phi, float* q) {
  // SO3 exponential map
  float theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  float theta_p4 = theta_sq * theta_sq;

  float theta = sqrtf(theta_sq);
  float imag, real;

  if (theta_sq < 1e-8) {
    imag = 0.5 - (1.0/48.0)*theta_sq + (1.0/3840.0)*theta_p4;
    real = 1.0 - (1.0/ 8.0)*theta_sq + (1.0/ 384.0)*theta_p4;
  } else {
    imag = sinf(0.5 * theta) / theta;
    real = cosf(0.5 * theta);
  }

  q[0] = imag * phi[0];
  q[1] = imag * phi[1];
  q[2] = imag * phi[2];
  q[3] = real;

}

__device__ void
crossInplace(const float* a, float *b) {
  float x[3] = {
    a[1]*b[2] - a[2]*b[1],
    a[2]*b[0] - a[0]*b[2],
    a[0]*b[1] - a[1]*b[0], 
  };

  b[0] = x[0];
  b[1] = x[1];
  b[2] = x[2];
}

__device__ void
expSE3(const float *xi, float* t, float* q) {
  // SE3 exponential map

  expSO3(xi + 3, q);
  float tau[3] = {xi[0], xi[1], xi[2]};
  float phi[3] = {xi[3], xi[4], xi[5]};

  float theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  float theta = sqrtf(theta_sq);

  t[0] = tau[0]; 
  t[1] = tau[1]; 
  t[2] = tau[2];

  if (theta > 1e-4) {
    float a = (1 - cosf(theta)) / theta_sq;
    crossInplace(phi, tau);
    t[0] += a * tau[0];
    t[1] += a * tau[1];
    t[2] += a * tau[2];

    float b = (theta - sinf(theta)) / (theta * theta_sq);
    crossInplace(phi, tau);
    t[0] += b * tau[0];
    t[1] += b * tau[1];
    t[2] += b * tau[2];
  }
}
__global__ void projective_transform_kernel(
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> target,
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> weight,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> jj,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> Hs,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> vs,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> Eii,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> Eij,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> Cii,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> bz)
{
  const int block_id = blockIdx.x;
  const int thread_id = threadIdx.x;

  const int ht = disps.size(1);
  const int wd = disps.size(2);

  int ix = static_cast<int>(ii[block_id]);
  int jx = static_cast<int>(jj[block_id]);

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  // load intrinsics from global memory
  if (thread_id == 0) {
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];
  }

  __syncthreads();

  // stereo frames
  if (ix == jx) {
    if (thread_id == 0) {
      tij[0] =  -0.1;
      tij[1] =     0;
      tij[2] =     0;
      qij[0] =     0;
      qij[1] =     0;
      qij[2] =     0;
      qij[3] =     1;
    }
  }

  else {

    // load poses from global memory
    if (thread_id < 3) {
      ti[thread_id] = poses[ix][thread_id];
      tj[thread_id] = poses[jx][thread_id];
    }

    if (thread_id < 4) {
      qi[thread_id] = poses[ix][thread_id+3];
      qj[thread_id] = poses[jx][thread_id+3];
    }

    __syncthreads();

    if (thread_id == 0) {
      relSE3(ti, qi, tj, qj, tij, qij);
    }
  }

  __syncthreads();

  //points 
  float Xi[4];
  float Xj[4];

  // jacobians
  float Jx[12];
  float Jz;

  float* Ji = &Jx[0];
  float* Jj = &Jx[6];

  // hessians
  float hij[12*(12+1)/2];

  float vi[6], vj[6];

  int l;
  for (l=0; l<12*(12+1)/2; l++) {
    hij[l] = 0;
  }

  for (int n=0; n<6; n++) {
    vi[n] = 0;
    vj[n] = 0;
  }

  __syncthreads();

  GPU_1D_KERNEL_LOOP(k, ht*wd) {

    const int i = k / wd;
    const int j = k % wd;

    const float u = static_cast<float>(j);
    const float v = static_cast<float>(i);
    
    // homogenous coordinates
    Xi[0] = (u - cx) / fx;
    Xi[1] = (v - cy) / fy;
    Xi[2] = 1;
    Xi[3] = disps[ix][i][j];

    // transform homogenous point
    actSE3(tij, qij, Xi, Xj);

    const float x = Xj[0];
    const float y = Xj[1];
    const float h = Xj[3];

    const float d = (Xj[2] < MIN_DEPTH) ? 0.0 : 1.0 / Xj[2];
    const float d2 = d * d;

    float wu = (Xj[2] < MIN_DEPTH) ? 0.0 : .001 * weight[block_id][0][i][j];
    float wv = (Xj[2] < MIN_DEPTH) ? 0.0 : .001 * weight[block_id][1][i][j];
    const float ru = target[block_id][0][i][j] - (fx * d * x + cx);
    const float rv = target[block_id][1][i][j] - (fy * d * y + cy);

    // x - coordinate

    Jj[0] = fx * (h*d);
    Jj[1] = fx * 0;
    Jj[2] = fx * (-x*h*d2);
    Jj[3] = fx * (-x*y*d2);
    Jj[4] = fx * (1 + x*x*d2);
    Jj[5] = fx * (-y*d);

    Jz = fx * (tij[0] * d - tij[2] * (x * d2));
    Cii[block_id][k] = wu * Jz * Jz;
    bz[block_id][k] = wu * ru * Jz;

    if (ix == jx) wu = 0;

    adjSE3(tij, qij, Jj, Ji);
    for (int n=0; n<6; n++) Ji[n] *= -1;

    l=0;
    for (int n=0; n<12; n++) {
      for (int m=0; m<=n; m++) {
        hij[l] += wu * Jx[n] * Jx[m];
        l++;
      }
    }

    for (int n=0; n<6; n++) {
      vi[n] += wu * ru * Ji[n];
      vj[n] += wu * ru * Jj[n];

      Eii[block_id][n][k] = wu * Jz * Ji[n];
      Eij[block_id][n][k] = wu * Jz * Jj[n];
    }


    Jj[0] = fy * 0;
    Jj[1] = fy * (h*d);
    Jj[2] = fy * (-y*h*d2);
    Jj[3] = fy * (-1 - y*y*d2);
    Jj[4] = fy * (x*y*d2);
    Jj[5] = fy * (x*d);

    Jz = fy * (tij[1] * d - tij[2] * (y * d2));
    Cii[block_id][k] += wv * Jz * Jz;
    bz[block_id][k] += wv * rv * Jz;

    if (ix == jx) wv = 0;

    adjSE3(tij, qij, Jj, Ji);
    for (int n=0; n<6; n++) Ji[n] *= -1;

    l=0;
    for (int n=0; n<12; n++) {
      for (int m=0; m<=n; m++) {
        hij[l] += wv * Jx[n] * Jx[m];
        l++;
      }
    }

    for (int n=0; n<6; n++) {
      vi[n] += wv * rv * Ji[n];
      vj[n] += wv * rv * Jj[n];

      Eii[block_id][n][k] += wv * Jz * Ji[n];
      Eij[block_id][n][k] += wv * Jz * Jj[n];
    }


  }

  __syncthreads();

  __shared__ float sdata[THREADS];
  for (int n=0; n<6; n++) {
    sdata[threadIdx.x] = vi[n];
    blockReduce(sdata);
    if (threadIdx.x == 0) {
      vs[0][block_id][n] = sdata[0];
    }

    __syncthreads();

    sdata[threadIdx.x] = vj[n];
    blockReduce(sdata);
    if (threadIdx.x == 0) {
      vs[1][block_id][n] = sdata[0];
    }

  }

  l=0;
  for (int n=0; n<12; n++) {
    for (int m=0; m<=n; m++) {
      sdata[threadIdx.x] = hij[l];
      blockReduce(sdata);

      if (threadIdx.x == 0) {
        if (n<6 && m<6) {
          Hs[0][block_id][n][m] = sdata[0];
          Hs[0][block_id][m][n] = sdata[0];
        }
        else if (n >=6 && m<6) {
          Hs[1][block_id][m][n-6] = sdata[0];
          Hs[2][block_id][n-6][m] = sdata[0];
        }
        else {
          Hs[3][block_id][n-6][m-6] = sdata[0];
          Hs[3][block_id][m-6][n-6] = sdata[0];
        }
      }

      l++;
    }
  }
}


__global__ void projmap_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> jj,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> coords,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> valid)
{

  const int block_id = blockIdx.x;
  const int thread_id = threadIdx.x;

  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ int ix;
  __shared__ int jx;

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  // load intrinsics from global memory
  if (thread_id == 0) {
    ix = static_cast<int>(ii[block_id]);
    jx = static_cast<int>(jj[block_id]);
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];
  }

  __syncthreads();

  // load poses from global memory
  if (thread_id < 3) {
    ti[thread_id] = poses[ix][thread_id];
    tj[thread_id] = poses[jx][thread_id];
  }

  if (thread_id < 4) {
    qi[thread_id] = poses[ix][thread_id+3];
    qj[thread_id] = poses[jx][thread_id+3];
  }

  __syncthreads();

  if (thread_id == 0) {
    relSE3(ti, qi, tj, qj, tij, qij);
  }

  //points 
  float Xi[4];
  float Xj[4];

  __syncthreads();

  GPU_1D_KERNEL_LOOP(k, ht*wd) {
    const int i = k / wd;
    const int j = k % wd;

    const float u = static_cast<float>(j);
    const float v = static_cast<float>(i);
    
    // homogenous coordinates
    Xi[0] = (u - cx) / fx;
    Xi[1] = (v - cy) / fy;
    Xi[2] = 1;
    Xi[3] = disps[ix][i][j];

    // transform homogenous point
    actSE3(tij, qij, Xi, Xj);

    coords[block_id][i][j][0] = u;
    coords[block_id][i][j][1] = v;

    if (Xj[2] > 0.01) {
      coords[block_id][i][j][0] = fx * (Xj[0] / Xj[2]) + cx;
      coords[block_id][i][j][1] = fy * (Xj[1] / Xj[2]) + cy;
    }

    valid[block_id][i][j][0] = (Xj[2] > MIN_DEPTH) ? 1.0 : 0.0;

  }
}

__global__ void frame_distance_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> jj,
    torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> dist,
    const float beta) {

  const int block_id = blockIdx.x;
  const int thread_id = threadIdx.x;

  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ int ix;
  __shared__ int jx;

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  // load intrinsics from global memory
  if (thread_id == 0) {
    ix = static_cast<int>(ii[block_id]);
    jx = static_cast<int>(jj[block_id]);
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];
  }

  __syncthreads();


  //points 
  float Xi[4];
  float Xj[4];

  __shared__ float accum[THREADS]; accum[thread_id] = 0;
  __shared__ float valid[THREADS]; valid[thread_id] = 0;
  __shared__ float total[THREADS]; total[thread_id] = 0;

  __syncthreads();

  for (int n=0; n<1; n++) {

    if (thread_id < 3) {
      ti[thread_id] = poses[ix][thread_id];
      tj[thread_id] = poses[jx][thread_id];
    }

    if (thread_id < 4) {
      qi[thread_id] = poses[ix][thread_id+3];
      qj[thread_id] = poses[jx][thread_id+3];
    }

    __syncthreads();


    relSE3(ti, qi, tj, qj, tij, qij);

    float d, du, dv;

    GPU_1D_KERNEL_LOOP(k, ht*wd) {
      const int i = k / wd;
      const int j = k % wd;

      const float u = static_cast<float>(j);
      const float v = static_cast<float>(i);


      // if (disps[ix][i][j] < 0.01) {
      //   continue;
      // }
      
      // homogenous coordinates
      Xi[0] = (u - cx) / fx;
      Xi[1] = (v - cy) / fy;
      Xi[2] = 1;
      Xi[3] = disps[ix][i][j];

      // transform homogenous point
      actSE3(tij, qij, Xi, Xj);

      du = fx * (Xj[0] / Xj[2]) + cx - u;
      dv = fy * (Xj[1] / Xj[2]) + cy - v;
      d = sqrtf(du*du + dv*dv);

      total[threadIdx.x] += beta;
      
      if (Xj[2] > MIN_DEPTH) {
        accum[threadIdx.x] += beta * d;
        valid[threadIdx.x] += beta;
      }

      Xi[0] = (u - cx) / fx;
      Xi[1] = (v - cy) / fy;
      Xi[2] = 1;
      Xi[3] = disps[ix][i][j];

      Xj[0] = Xi[0] + Xi[3] * tij[0];
      Xj[1] = Xi[1] + Xi[3] * tij[1];
      Xj[2] = Xi[2] + Xi[3] * tij[2];

      du = fx * (Xj[0] / Xj[2]) + cx - u;
      dv = fy * (Xj[1] / Xj[2]) + cy - v;
      d = sqrtf(du*du + dv*dv);

      total[threadIdx.x] += (1 - beta);
      
      if (Xj[2] > MIN_DEPTH) {
        accum[threadIdx.x] += (1 - beta) * d;
        valid[threadIdx.x] += (1 - beta);
      }
    }

    if (threadIdx.x == 0) {
      int tmp = ix;
      ix = jx;
      jx = tmp;
    }

    __syncthreads();

  }
  __syncthreads(); blockReduce(accum);
  __syncthreads(); blockReduce(total);
  __syncthreads(); blockReduce(valid);

  __syncthreads();

  if (thread_id == 0) {
    dist[block_id] = (valid[0] / (total[0] + 1e-8) < 0.75) ? 1000.0 : accum[0] / valid[0];
  }
}



__global__ void depth_filter_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> inds,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> thresh,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> counter)
{

  const int block_id = blockIdx.x;
  const int neigh_id = blockIdx.y;
  const int index = blockIdx.z * blockDim.x + threadIdx.x;

  // if (threadIdx.x == 0) {
  //   printf("%d %d %d %d\n", blockIdx.x, blockIdx.y, blockDim.x, threadIdx.x);
  // }

  const int num = disps.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ int ix;
  __shared__ int jx;

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;

  __shared__ float ti[3], tj[3], tij[3];
  __shared__ float qi[4], qj[4], qij[4];

  if (threadIdx.x == 0) {
    ix = static_cast<int>(inds[block_id]);
    jx = (neigh_id < 3) ? ix - neigh_id - 1 : ix + neigh_id;
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];
  }

  __syncthreads();

  if (jx < 0 || jx >= num) {
    return;
  }

  const float t = thresh[block_id];

  // load poses from global memory
  if (threadIdx.x < 3) {
    ti[threadIdx.x] = poses[ix][threadIdx.x];
    tj[threadIdx.x] = poses[jx][threadIdx.x];
  }

  if (threadIdx.x < 4) {
    qi[threadIdx.x] = poses[ix][threadIdx.x+3];
    qj[threadIdx.x] = poses[jx][threadIdx.x+3];
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    relSE3(ti, qi, tj, qj, tij, qij);
  }

  //points 
  float Xi[4];
  float Xj[4];

  __syncthreads();

  if (index < ht*wd) {
    const int i = index / wd;
    const int j = index % wd;

    const float ui = static_cast<float>(j);
    const float vi = static_cast<float>(i);
    const float di = disps[ix][i][j];
    
    // homogenous coordinates
    Xi[0] = (ui - cx) / fx;
    Xi[1] = (vi - cy) / fy;
    Xi[2] = 1;
    Xi[3] = di;

    // transform homogenous point
    actSE3(tij, qij, Xi, Xj);

    const float uj = fx * (Xj[0] / Xj[2]) + cx;
    const float vj = fy * (Xj[1] / Xj[2]) + cy;
    const float dj = Xj[3] / Xj[2];

    const int u0 = static_cast<int>(floor(uj));
    const int v0 = static_cast<int>(floor(vj));

    if (u0 >= 0 && v0 >= 0 && u0 < wd-1 && v0 < ht-1) {
      const float wx = ceil(uj) - uj;
      const float wy = ceil(vj) - vj;

      const float d00 = disps[jx][v0+0][u0+0];
      const float d01 = disps[jx][v0+0][u0+1];
      const float d10 = disps[jx][v0+1][u0+0];
      const float d11 = disps[jx][v0+1][u0+1];

      const float dj_hat = wy*wx*d00 + wy*(1-wx)*d01 + (1-wy)*wx*d10 + (1-wy)*(1-wx)*d11;

      const float err = abs(1.0/dj - 1.0/dj_hat);
      if       (abs(1.0/dj - 1.0/d00) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
      else if  (abs(1.0/dj - 1.0/d01) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
      else if  (abs(1.0/dj - 1.0/d10) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
      else if  (abs(1.0/dj - 1.0/d11) < t) atomicAdd(&counter[block_id][i][j], 1.0f);
    }
  }
}



__global__ void iproj_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,1,torch::RestrictPtrTraits> intrinsics,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> points)

{

  const int block_id = blockIdx.x;
  const int index = blockIdx.y * blockDim.x + threadIdx.x;


  const int num = disps.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  __shared__ float fx;
  __shared__ float fy;
  __shared__ float cx;
  __shared__ float cy;

  __shared__ float t[3];
  __shared__ float q[4];

  if (threadIdx.x == 0) {
    fx = intrinsics[0];
    fy = intrinsics[1];
    cx = intrinsics[2];
    cy = intrinsics[3];
  }

  __syncthreads();


  // load poses from global memory
  if (threadIdx.x < 3) {
    t[threadIdx.x] = poses[block_id][threadIdx.x];
  }

  if (threadIdx.x < 4) {
    q[threadIdx.x] = poses[block_id][threadIdx.x+3];
  }

  __syncthreads();

  //points 
  float Xi[4];
  float Xj[4];

  if (index < ht*wd) {
    const int i = index / wd;
    const int j = index % wd;

    const float ui = static_cast<float>(j);
    const float vi = static_cast<float>(i);
    const float di = disps[block_id][i][j];
    
    // homogenous coordinates
    Xi[0] = (ui - cx) / fx;
    Xi[1] = (vi - cy) / fy;
    Xi[2] = 1;
    Xi[3] = di;

    // transform homogenous point
    actSE3(t, q, Xi, Xj);

    points[block_id][i][j][0] = Xj[0] / Xj[3];
    points[block_id][i][j][1] = Xj[1] / Xj[3];
    points[block_id][i][j][2] = Xj[2] / Xj[3];

  }
}



__global__ void accum_kernel(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> inps,
    const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> ptrs,
    const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> idxs,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> outs)
{
  
  const int block_id = blockIdx.x;
  const int D = inps.size(2);

  const int start = ptrs[block_id];
  const int end = ptrs[block_id+1];

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    float x = 0;
    for (int i=start; i<end; i++) {
      x += inps[idxs[i]][k];
    }
    outs[block_id][k] = x;
  }  
}


__device__ void
retrSE3(const float *xi, const float* t, const float* q, float* t1, float* q1) {
  // retraction on SE3 manifold

  float dt[3] = {0, 0, 0};
  float dq[4] = {0, 0, 0, 1};
  
  expSE3(xi, dt, dq);

  q1[0] = dq[3] * q[0] + dq[0] * q[3] + dq[1] * q[2] - dq[2] * q[1];
  q1[1] = dq[3] * q[1] + dq[1] * q[3] + dq[2] * q[0] - dq[0] * q[2];
  q1[2] = dq[3] * q[2] + dq[2] * q[3] + dq[0] * q[1] - dq[1] * q[0];
  q1[3] = dq[3] * q[3] - dq[0] * q[0] - dq[1] * q[1] - dq[2] * q[2];

  actSO3(dq, t, t1);
  t1[0] += dt[0];
  t1[1] += dt[1];
  t1[2] += dt[2];
}


__global__ void pose_retr_kernel(
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> dx,
    const int t0, const int t1) 
{

  for (int k=t0+threadIdx.x; k<t1; k+=blockDim.x) {
    float xi[6], q[4], q1[4], t[3], t1[3];

    t[0] = poses[k][0];
    t[1] = poses[k][1];
    t[2] = poses[k][2];

    q[0] = poses[k][3];
    q[1] = poses[k][4];
    q[2] = poses[k][5];
    q[3] = poses[k][6];
    
    for (int n=0; n<6; n++) {
      xi[n] = dx[k-t0][n];
    }

    retrSE3(xi, t, q, t1, q1);

    poses[k][0] = t1[0];
    poses[k][1] = t1[1];
    poses[k][2] = t1[2];

    poses[k][3] = q1[0];
    poses[k][4] = q1[1];
    poses[k][5] = q1[2];
    poses[k][6] = q1[3];
  }
}

__global__ void disp_retr_kernel(
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> disps,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> dz,
    const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> inds) 
{
  const int i = inds[blockIdx.x];
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  for (int k=threadIdx.x; k<ht*wd; k+=blockDim.x) {
    float d = disps[i][k/wd][k%wd] + dz[blockIdx.x][k];
    disps[i][k/wd][k%wd] = d;
  }
}

torch::Tensor accum_cuda(torch::Tensor data, torch::Tensor ix, torch::Tensor jx) {
  torch::Tensor ix_cpu = ix.to(torch::kCPU);
  torch::Tensor jx_cpu = jx.to(torch::kCPU);
  torch::Tensor inds = torch::argsort(ix_cpu);

  LongType* ix_data = ix_cpu.data_ptr<LongType>();
  LongType* jx_data = jx_cpu.data_ptr<LongType>();
  LongType* kx_data = inds.data_ptr<LongType>();

  int count = jx.size(0);
  std::vector<int> cols;

  torch::Tensor ptrs_cpu = torch::zeros({count+1}, 
    torch::TensorOptions().dtype(torch::kInt64));
  
  LongType* ptrs_data = ptrs_cpu.data_ptr<LongType>();
  ptrs_data[0] = 0;

  int i = 0;
  for (int j=0; j<count; j++) {
    while (i < ix.size(0) && ix_data[kx_data[i]] <= jx_data[j]) {
      if (ix_data[kx_data[i]] == jx_data[j])
        cols.push_back(kx_data[i]);
      i++;
    }
    ptrs_data[j+1] = cols.size();
  }

  torch::Tensor idxs_cpu = torch::zeros({LongType(cols.size())}, 
    torch::TensorOptions().dtype(torch::kInt64));

  LongType* idxs_data = idxs_cpu.data_ptr<LongType>();

  for (int i=0; i<cols.size(); i++) {
    idxs_data[i] = cols[i];
  }

  torch::Tensor ptrs = ptrs_cpu.to(torch::kCUDA);
  torch::Tensor idxs = idxs_cpu.to(torch::kCUDA);

  torch::Tensor out = torch::zeros({jx.size(0), data.size(1)},
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

  accum_kernel<<<count, THREADS>>>(
    data.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    ptrs.packed_accessor32<LongType,1,torch::RestrictPtrTraits>(),
    idxs.packed_accessor32<LongType,1,torch::RestrictPtrTraits>(),
    out.packed_accessor32<float,2,torch::RestrictPtrTraits>());

  return out;
}


__global__ void EEt6x6_kernel(
    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> E,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> Q,
    const torch::PackedTensorAccessor32<LongType,2,torch::RestrictPtrTraits> idx,
    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> S)
{

  // indicices
  const int ix = idx[blockIdx.x][0];
  const int jx = idx[blockIdx.x][1];
  const int kx = idx[blockIdx.x][2];

  const int D = E.size(2);

  float dS[6][6];
  float ei[6];
  float ej[6];

  for (int i=0; i<6; i++) {
    for (int j=0; j<6; j++) {
      dS[i][j] = 0;
    }
  }

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    const float q = Q[kx][k];
      
    // coalesced memory read
    for (int n=0; n<6; n++) {
      ei[n] = E[ix][n][k] * q;
      ej[n] = E[jx][n][k];
    }

    // block EEt
    for (int n=0; n<6; n++) {
      for (int m=0; m<6; m++) {
        dS[n][m] += ei[n] * ej[m];
      }
    }
  }

  __syncthreads();
  __shared__ float sdata[THREADS];

  for (int n=0; n<6; n++) {
    for (int m=0; m<6; m++) {
      sdata[threadIdx.x] = dS[n][m];

      blockReduce(sdata);

      if (threadIdx.x == 0) {
        S[blockIdx.x][n][m] = sdata[0];
      }
    }
  }
}


__global__ void Ev6x1_kernel(
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> E,
    const torch::PackedTensorAccessor32<float, 2,torch::RestrictPtrTraits> Q,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> w,
    const torch::PackedTensorAccessor32<LongType,2,torch::RestrictPtrTraits> idx,
    torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> v)
{
  const int D = E.size(2);
  const int kx = idx[blockIdx.x][0];

  float b[6];
  for (int n=0; n<6; n++) {
    b[n] = 0.0;
  }

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    const float q_w = Q[kx][k] * w[kx][k];

    for (int n=0; n<6; n++) {
      b[n] += q_w * E[blockIdx.x][n][k];
    }
  }

  __syncthreads();
  __shared__ float sdata[THREADS];

  for (int n=0; n<6; n++) {
    sdata[threadIdx.x] = b[n];
    blockReduce(sdata);

    if (threadIdx.x == 0) {
      v[blockIdx.x][n] += sdata[0];
    }
  }
}

__global__ void EvT6x1_kernel(
  const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> E,
  const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> x,
  const torch::PackedTensorAccessor32<LongType,1,torch::RestrictPtrTraits> idx,
  torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> w)
{

  const int D = E.size(2);
  const int ix = idx[blockIdx.x];

  if (idx[blockIdx.x] <= 0 || idx[blockIdx.x] >= x.size(0))
    return;

  for (int k=threadIdx.x; k<D; k+=blockDim.x) {
    float dw = 0;
    for (int n=0; n<6; n++) {
      dw += E[blockIdx.x][n][k] * x[ix][n];
    }
    w[blockIdx.x][k] = dw;
  }
}

class SparseBlock {
  public:

    Eigen::SparseMatrix<double> A;
    Eigen::VectorX<double> b;

    SparseBlock(int N, int M) : N(N), M(M) {
      A = Eigen::SparseMatrix<double>(N*M, N*M);
      b = Eigen::VectorXd::Zero(N*M);
    }

    SparseBlock(Eigen::SparseMatrix<double> const& A, Eigen::VectorX<double> const& b, 
        int N, int M) : A(A), b(b), N(N), M(M) {}

    void update_lhs(torch::Tensor As, torch::Tensor ii, torch::Tensor jj) {

      auto As_cpu = As.to(torch::kCPU).to(torch::kFloat64);
      auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);
      auto jj_cpu = jj.to(torch::kCPU).to(torch::kInt64);

      auto As_acc = As_cpu.accessor<double,3>();
      auto ii_acc = ii_cpu.accessor<LongType,1>();
      auto jj_acc = jj_cpu.accessor<LongType,1>();

      std::vector<T> tripletList;
      for (int n=0; n<ii.size(0); n++) {
        const int i = ii_acc[n];
        const int j = jj_acc[n];

        if (i >= 0 && j >= 0) {
          for (int k=0; k<M; k++) {
            for (int l=0; l<M; l++) {
              double val = As_acc[n][k][l];
              tripletList.push_back(T(M*i + k, M*j + l, val));
            }
          }
        }
      }
      A.setFromTriplets(tripletList.begin(), tripletList.end());
    }

    void update_rhs(torch::Tensor bs, torch::Tensor ii) {
      auto bs_cpu = bs.to(torch::kCPU).to(torch::kFloat64);
      auto ii_cpu = ii.to(torch::kCPU).to(torch::kInt64);

      auto bs_acc = bs_cpu.accessor<double,2>();
      auto ii_acc = ii_cpu.accessor<LongType,1>();

      for (int n=0; n<ii.size(0); n++) {
        const int i = ii_acc[n];
        if (i >= 0) {
          for (int j=0; j<M; j++) {
            b(i*M + j) += bs_acc[n][j];
          }
        }
      }
    }

    SparseBlock operator-(const SparseBlock& S) {
      return SparseBlock(A - S.A, b - S.b, N, M);
    }

    std::tuple<torch::Tensor, torch::Tensor> get_dense() {
      Eigen::MatrixXd Ad = Eigen::MatrixXd(A);

      torch::Tensor H = torch::from_blob(Ad.data(), {N*M, N*M}, torch::TensorOptions()
        .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);

      torch::Tensor v = torch::from_blob(b.data(), {N*M, 1}, torch::TensorOptions()
        .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);

      return std::make_tuple(H, v);

    }

    torch::Tensor solve(const float lm=0.0001, const float ep=0.1) {

      torch::Tensor dx;

      Eigen::SparseMatrix<double> L(A);
      L.diagonal().array() += ep + lm * L.diagonal().array();

      Eigen::SimplicialLLT<Eigen::SparseMatrix<double>> solver;
      solver.compute(L);

      if (solver.info() == Eigen::Success) {
        Eigen::VectorXd x = solver.solve(b);
        dx = torch::from_blob(x.data(), {N, M}, torch::TensorOptions()
          .dtype(torch::kFloat64)).to(torch::kCUDA).to(torch::kFloat32);
      }
      else {
        dx = torch::zeros({N, M}, torch::TensorOptions()
          .device(torch::kCUDA).dtype(torch::kFloat32));
      }
      
      return dx;
    }

  private:
    const int N;
    const int M;

};


SparseBlock schur_block(torch::Tensor E,
                        torch::Tensor Q,
                        torch::Tensor w,
                        torch::Tensor ii,
                        torch::Tensor jj,
                        torch::Tensor kk,
                        const int t0,
                        const int t1)
{

  torch::Tensor ii_cpu = ii.to(torch::kCPU);
  torch::Tensor jj_cpu = jj.to(torch::kCPU);
  torch::Tensor kk_cpu = kk.to(torch::kCPU);

  const int P = t1 - t0;
  const LongType* ii_data = ii_cpu.data_ptr<LongType>();
  const LongType* jj_data = jj_cpu.data_ptr<LongType>();
  const LongType* kk_data = kk_cpu.data_ptr<LongType>();

  std::vector<std::vector<LongType>> graph(P);
  std::vector<std::vector<LongType>> index(P);

  for (int n=0; n<ii_cpu.size(0); n++) {
    const int j = jj_data[n];
    const int k = kk_data[n];

    if (j >= t0 && j <= t1) {
      const int t = j - t0;
      graph[t].push_back(k);
      index[t].push_back(n);
    }
  }

  std::vector<LongType> ii_list, jj_list, idx, jdx;

  for (int i=0; i<P; i++) {
    for (int j=0; j<P; j++) {
      for (int k=0; k < graph[i].size(); k++) {
        for (int l=0; l < graph[j].size(); l++) {
          if (graph[i][k] == graph[j][l]) {
            ii_list.push_back(i);
            jj_list.push_back(j);

            idx.push_back(index[i][k]);
            idx.push_back(index[j][l]);
            idx.push_back(graph[i][k]);
          }
        }
      }
    }
  }

  torch::Tensor ix_cuda = torch::from_blob(idx.data(), {LongType(idx.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).to(torch::kCUDA).view({-1, 3});

  torch::Tensor jx_cuda = torch::stack({kk_cpu}, -1)
    .to(torch::kCUDA).to(torch::kInt64);

  torch::Tensor ii2_cpu = torch::from_blob(ii_list.data(), {LongType(ii_list.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).view({-1});

  torch::Tensor jj2_cpu = torch::from_blob(jj_list.data(), {LongType(jj_list.size())}, 
    torch::TensorOptions().dtype(torch::kInt64)).view({-1});

  torch::Tensor S = torch::zeros({ix_cuda.size(0), 6, 6}, 
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

  torch::Tensor v = torch::zeros({jx_cuda.size(0), 6},
    torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA));

  EEt6x6_kernel<<<ix_cuda.size(0), THREADS>>>(
    E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    ix_cuda.packed_accessor32<LongType,2,torch::RestrictPtrTraits>(),
    S.packed_accessor32<float,3,torch::RestrictPtrTraits>());

  Ev6x1_kernel<<<jx_cuda.size(0), THREADS>>>(
    E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    Q.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    w.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    jx_cuda.packed_accessor32<LongType,2,torch::RestrictPtrTraits>(),
    v.packed_accessor32<float,2,torch::RestrictPtrTraits>());

  // schur block
  SparseBlock A(P, 6);
  A.update_lhs(S, ii2_cpu, jj2_cpu);
  A.update_rhs(v, jj_cpu - t0);

  return A;
}


std::vector<torch::Tensor> ba_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor disps_sens,
    torch::Tensor targets,
    torch::Tensor weights,
    torch::Tensor eta,
    torch::Tensor ii,
    torch::Tensor jj,
    const int t0,
    const int t1,
    const int iterations,
    const float lm,
    const float ep,
    const bool motion_only)
{
  auto opts = poses.options();
  const int num = ii.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  torch::Tensor ts = torch::arange(t0, t1).to(torch::kCUDA);
  torch::Tensor ii_exp = torch::cat({ts, ii}, 0);
  torch::Tensor jj_exp = torch::cat({ts, jj}, 0);

  std::tuple<torch::Tensor, torch::Tensor> kuniq = 
    torch::_unique(ii_exp, true, true);

  torch::Tensor kx = std::get<0>(kuniq);
  torch::Tensor kk_exp = std::get<1>(kuniq);
    
  torch::Tensor dx;
  torch::Tensor dz;

  // initialize buffers
  torch::Tensor Hs = torch::zeros({4, num, 6, 6}, opts);
  torch::Tensor vs = torch::zeros({2, num, 6}, opts);
  torch::Tensor Eii = torch::zeros({num, 6, ht*wd}, opts);
  torch::Tensor Eij = torch::zeros({num, 6, ht*wd}, opts);
  torch::Tensor Cii = torch::zeros({num, ht*wd}, opts);
  torch::Tensor wi = torch::zeros({num, ht*wd}, opts);

  for (int itr=0; itr<iterations; itr++) {

    projective_transform_kernel<<<num, THREADS>>>(
      targets.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      weights.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
      ii.packed_accessor32<LongType,1,torch::RestrictPtrTraits>(),
      jj.packed_accessor32<LongType,1,torch::RestrictPtrTraits>(),
      Hs.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
      vs.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      Eii.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      Eij.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
      Cii.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
      wi.packed_accessor32<float,2,torch::RestrictPtrTraits>());


    // pose x pose block
    SparseBlock A(t1 - t0, 6);

    A.update_lhs(Hs.reshape({-1, 6, 6}), 
        torch::cat({ii, ii, jj, jj}) - t0, 
        torch::cat({ii, jj, ii, jj}) - t0);

    A.update_rhs(vs.reshape({-1, 6}), 
        torch::cat({ii, jj}) - t0);

    if (motion_only) {
      dx = A.solve(lm, ep);

      // update poses
      pose_retr_kernel<<<1, THREADS>>>(
        poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(), t0, t1);
    }
    
    else {
      // add depth residual if there are depth sensor measurements
      const float alpha = 0.05;
      torch::Tensor m = (disps_sens.index({kx, "..."}) > 0).to(torch::TensorOptions().dtype(torch::kFloat32)).view({-1, ht*wd});
      torch::Tensor C = accum_cuda(Cii, ii, kx) + m * alpha + (1 - m) * eta.view({-1, ht*wd});
      torch::Tensor w = accum_cuda(wi, ii, kx) - m * alpha * (disps.index({kx, "..."}) - disps_sens.index({kx, "..."})).view({-1, ht*wd});
      torch::Tensor Q = 1.0 / C;

      torch::Tensor Ei = accum_cuda(Eii.view({num, 6*ht*wd}), ii, ts).view({t1-t0, 6, ht*wd});
      torch::Tensor E = torch::cat({Ei, Eij}, 0);

      SparseBlock S = schur_block(E, Q, w, ii_exp, jj_exp, kk_exp, t0, t1);
      dx = (A - S).solve(lm, ep);

      torch::Tensor ix = jj_exp - t0;
      torch::Tensor dw = torch::zeros({ix.size(0), ht*wd}, opts);

      EvT6x1_kernel<<<ix.size(0), THREADS>>>(
        E.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
        dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        ix.packed_accessor32<LongType,1,torch::RestrictPtrTraits>(),
        dw.packed_accessor32<float,2,torch::RestrictPtrTraits>());

      dz = Q * (w - accum_cuda(dw, ii_exp, kx));

      // update poses
      pose_retr_kernel<<<1, THREADS>>>(
        poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        dx.packed_accessor32<float,2,torch::RestrictPtrTraits>(), t0, t1);

      // update disparity maps
      disp_retr_kernel<<<kx.size(0), THREADS>>>(
        disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
        dz.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
        kx.packed_accessor32<LongType,1,torch::RestrictPtrTraits>());
    }

  }

  return {dx, dz};
}



torch::Tensor frame_distance_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor ii,
    torch::Tensor jj,
    const float beta)
{
  auto opts = poses.options();
  const int num = ii.size(0);

  torch::Tensor dist = torch::zeros({num}, opts);

  frame_distance_kernel<<<num, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    ii.packed_accessor32<LongType,1,torch::RestrictPtrTraits>(),
    jj.packed_accessor32<LongType,1,torch::RestrictPtrTraits>(),
    dist.packed_accessor32<float,1,torch::RestrictPtrTraits>(), beta);

  return dist;
}


std::vector<torch::Tensor> projmap_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor ii,
    torch::Tensor jj)
{
  auto opts = poses.options();
  const int num = ii.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  torch::Tensor coords = torch::zeros({num, ht, wd, 3}, opts);
  torch::Tensor valid = torch::zeros({num, ht, wd, 1}, opts);

  projmap_kernel<<<num, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    ii.packed_accessor32<LongType,1,torch::RestrictPtrTraits>(),
    jj.packed_accessor32<LongType,1,torch::RestrictPtrTraits>(),
    coords.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    valid.packed_accessor32<float,4,torch::RestrictPtrTraits>());

  return {coords, valid};
}


torch::Tensor depth_filter_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics,
    torch::Tensor ix,
    torch::Tensor thresh)
{
  const int num = ix.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  torch::Tensor counter = torch::zeros({num, ht, wd}, disps.options());

  dim3 blocks(num, 6, NUM_BLOCKS(ht * wd));

  depth_filter_kernel<<<blocks, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    ix.packed_accessor32<LongType,1,torch::RestrictPtrTraits>(),
    thresh.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    counter.packed_accessor32<float,3,torch::RestrictPtrTraits>());

  return counter;
}


torch::Tensor iproj_cuda(
    torch::Tensor poses,
    torch::Tensor disps,
    torch::Tensor intrinsics)
{

  const int nm = disps.size(0);
  const int ht = disps.size(1);
  const int wd = disps.size(2);

  auto opts = disps.options();
  torch::Tensor points = torch::zeros({nm, ht, wd, 3}, opts);

  dim3 blocks(nm, NUM_BLOCKS(ht * wd));

  iproj_kernel<<<blocks, THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    disps.packed_accessor32<float,3,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,1,torch::RestrictPtrTraits>(),
    points.packed_accessor32<float,4,torch::RestrictPtrTraits>());

  return points;

}
